#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace ORB_CUDA { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
