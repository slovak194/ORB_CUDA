#include <hip/hip_runtime_api.h>
#include <orb_cuda/cuda/Cuda.hpp>

namespace ORB_CUDA { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
